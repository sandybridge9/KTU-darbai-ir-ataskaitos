#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <string>
#include <iostream>
#include <iomanip>
#include <fstream>
#include <vector>
#include <locale>
#include <algorithm>
#include <cstdio>
//-------------------------------------------------
//#ifndef __HIPCC__ 
//#define __HIPCC__
//#endif
//#include "hip/hip_runtime.h"
//#include ""
//#include <hip/hip_runtime.h>
//#include <hip/device_functions.h>
//#include <hip/hip_runtime_api.h>

const int maxWordLength = 10;
const int arrayCount = 5;
const int inArrayCount = 11;
//std::vector<std::vector<Stud>> Read(std::string fileName);

//__device__ void charCpy(int index, char *dataWord, char *name);
//__global__ void addKernel(Stud **Q, Stud *Ans);

class Stud {
public:
	char name[maxWordLength * arrayCount] = {};
	int grades;
	double average;
	Stud() {}
	Stud(char name[], int grades, double average) {
		strcpy(this->name, name);
		this->grades = grades;
		this->average = average;
	}
};

__device__ void charCpy(int index, char *dataWord, char *name) {
	int tempi = 0;
	while (dataWord[tempi] != NULL)
	{
		name[index] = dataWord[tempi];
		tempi++;
		index++;
	}
}

__global__ void addKernel(Stud **Q, Stud *Ans)
{
	int i = threadIdx.x;
	
	__shared__ char name[inArrayCount][arrayCount * maxWordLength];
	__shared__ int nmb[inArrayCount];
	__shared__ double average[inArrayCount];
	
	nmb[i] = 0;
	average[i] = 0;
	for (int j = 0; j < arrayCount * maxWordLength; j++)
	{
		name[i][j] = NULL;
	}

	for (int j = 0; j < arrayCount; j++)
	{
		for (int q = 0; q < maxWordLength * arrayCount; q++)
		{
			if (name[i][q] == NULL)
			{
				charCpy(q, Q[j][i].name, name[i]);
				break;
			}
		}
		nmb[i] += Q[j][i].grades;
		average[i] = average[i] + Q[j][i].average;
	}

	charCpy(0, name[i], Ans[i].name);
	Ans[i].grades = nmb[i];
	Ans[i].average = average[i];
	//printf("%d-%s", i, Ans[i].name);
}



std::vector<std::vector<Stud>> Read(std::string fileName/*, std::vector<std::vector<Stud>> A*/)
{
	std::vector<std::vector<Stud>> A;
	std::ifstream in;
	in.open(fileName);
	while (!in.eof())
	{

		int len;
		in >> len;
		std::vector<Stud> temp;
		for (size_t i = 0; i < len; i++)
		{
			std::string name;
			in >> name;
			int nmb;
			in >> nmb;
			double average;
			in >> average;
			char tempWord[maxWordLength] = {};
			std::transform(name.begin(), name.end(), name.begin(), ::tolower);//change string letters to lower cases
			strcpy(tempWord, name.c_str());//put string to chars
			temp.push_back(Stud(tempWord, nmb, average));
		}
		A.push_back(temp);
	}
	in.close();
	return A;
}

void Write(std::string fileName, std::vector<std::vector<Stud>> dataArray, Stud *Ans)
{
	std::ofstream out;
	out.open(fileName);

	int a = 0;
	out << "Pradiniai duomenys" << std::endl;
	for (std::vector<Stud> data : dataArray)
	{
		//out << data[a].getLessonName() << endl;
		out << "Nr. Vardas    Pazymiu_skaicius  Vidurkis" << std::endl;
		int s = 0;
		for (Stud student : data)
		{
			s++;
			out << std::left << std::setw(4) << s << std::setw(10) << student.name << std::setw(18) << student.grades << student.average << std::endl;
		}
		a++;
	}
	out << std::endl;
	out << "Rezultatas" << std::endl;
	out << std::left << std::setw(40) << "Vardai" << std::setw(10) << "Pazymiai" << "Vidurkiai" << std::endl;
	for (int i = 0; i < inArrayCount; i++)
	{
		out << std::left << std::setw(40) << Ans[i].name << std::setw(5) << Ans[i].grades << Ans[i].average << std::endl;
	}
	out << std::endl;
	out.close();
}

int main()
{
	std::vector<std::vector<Stud>> A;//data
	/*const int arrayCount = 5;
	const int inArrayCount = 11;*/

	A = Read("IFF68_LaurinaitisTadas_L4.txt");
	
	std::vector<std::vector<Stud>> dataArray;//data
	dataArray = A;
	Stud arrayA[arrayCount][inArrayCount];
	for (int i = 0; i < arrayCount; i++)
	{
		for (int j = 0; j < inArrayCount; j++)
		{
			arrayA[i][j] = A[i][j];
		}
	}
	//dataArray = arrayA;

	Stud** dev_Q;
	hipMalloc((void**)&dev_Q, arrayCount * sizeof(Stud*));//for stud arrays 5
	for (int i = 0; i < arrayCount; i++)
	{
		Stud* temp_Q = nullptr;
		hipMalloc((void**)&temp_Q, inArrayCount * sizeof(Stud));
		hipMemcpy(temp_Q, &arrayA[i], inArrayCount * sizeof(Stud), hipMemcpyHostToDevice);
		hipMemcpy(&dev_Q[i], &temp_Q, sizeof(Stud*), hipMemcpyHostToDevice);
	}
	Stud Ans[inArrayCount] = {};
	Stud *dev_Ans;
	hipMalloc((void**)&dev_Ans, inArrayCount * sizeof(Stud));
	hipMemcpy(dev_Ans, Ans, inArrayCount * sizeof(int), hipMemcpyHostToDevice);
	//addKernel << < 1, arrayCount >> > (dev_Q, dev_Ans);
	addKernel << < 1, inArrayCount >> > (dev_Q, dev_Ans);
	hipMemcpy(Ans, dev_Ans, inArrayCount * sizeof(Stud), hipMemcpyDeviceToHost);
	hipFree(dev_Q);
	hipFree(dev_Ans);

	for (int i = 0; i < inArrayCount; i++)
	{
		std::cout << Ans[i].name << " " << Ans[i].grades << " " << Ans[i].average << " ";
		std::cout << std::endl;
	}
	Write("IFF68_LaurinaitisTadas_L4a_rez.txt", dataArray, Ans);
    return 0;
}