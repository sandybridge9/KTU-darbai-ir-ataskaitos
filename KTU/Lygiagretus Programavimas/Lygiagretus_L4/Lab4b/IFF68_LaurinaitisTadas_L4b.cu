#include "hip/hip_runtime.h"
#include ""

#include <string>
#include <iostream>
#include <iomanip>
#include <fstream>
#include <vector>
#include <locale>
#include <algorithm>
#include <cstdio>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

using namespace std;

const int maxWordLength = 10;
const int arrayCount = 5;
const int inArrayCount = 11;

class Stud {
public:
	char name[maxWordLength * arrayCount] = {};
	int grades = 0;
	double average = 0;
	__host__ __device__ Stud() {}
	__host__ __device__ Stud(char name[], int grades, double average) {
		for (int i = 0; i < maxWordLength * arrayCount; i++) {
			this->name[i] = name[i];
			if (name[i] == NULL)
			{
				break;
			}
		}
		//strcpy(this->name, name);
		this->grades = grades;
		this->average = average;
	}
};

//struct is_good {
//	const int index;
//
//	is_good(int _index) : index(_index) {}
//
//	__device__
//		bool operator()() {
//		return index % inArrayCount == 0;
//	}
//	/*__device__ bool operator ()(int index) {
//		return index % inArrayCount == 0;
//	}*/
//};

struct sum_func {
	__device__ Stud operator ()(Stud accumulator, Stud item) {
		for (int i = 0; i < maxWordLength * arrayCount; i++) {
			if (accumulator.name[i] == NULL) {
				int index = i;
				int tempi = 0;
				while (item.name[tempi] != NULL)
				{
					accumulator.name[index] = item.name[tempi];
					tempi++;
					index++;
				}
				break;
			}
		}
		accumulator.grades += item.grades;
		accumulator.average += item.average;
		return accumulator;
	}
};

thrust::host_vector<Stud> Read(std::string fileName)
{
	//thrust::host_vector<thrust::host_vector<Stud>> A;
	thrust::host_vector<Stud> A;
	std::ifstream in;
	in.open(fileName);
	while (!in.eof())
	{

		int len;
		in >> len;
		//thrust::host_vector<Stud> temp;
		//std::vector<Stud> temp;
		for (size_t i = 0; i < len; i++)
		{
			std::string name;
			in >> name;
			int nmb;
			in >> nmb;
			double average;
			in >> average;
			char tempWord[maxWordLength] = {};
			std::transform(name.begin(), name.end(), name.begin(), ::tolower);//change string letters to lower cases
			strcpy(tempWord, name.c_str());//put string to chars
			A.push_back(Stud(tempWord, nmb, average));
		}
		//A.push_back(temp);
	}
	in.close();
	return A;
}

void Write(std::string fileName, thrust::host_vector<Stud> dataArray, thrust::host_vector<Stud> Ans)
{
	std::ofstream out;
	out.open(fileName);

	int a = 0;
	out << "Pradiniai duomenys" << std::endl;
	for (int i = 0; i < arrayCount; i++)//5
	{
		out << "Nr. Vardas    Pazymiu_skaicius  Vidurkis" << std::endl;
		int s = 0;
		for (int j = 0; j < inArrayCount; j++)//11
		{
			out << std::left << std::setw(4) << j + 1 << std::setw(10) << static_cast<Stud>(dataArray[j]).name << std::setw(18) << static_cast<Stud>(dataArray[j]).grades << static_cast<Stud>(dataArray[j]).average << std::endl;
		}
	}
	out << std::endl;
	out << "Rezultatas" << std::endl;
	out << std::left << std::setw(40) << "Vardai" << std::setw(10) << "Pazymiai" << "Vidurkiai" << std::endl;
	for (int i = 0; i < inArrayCount; i++)
	{
		out << std::left << std::setw(40) << static_cast<Stud>(Ans[i]).name << std::setw(10) << static_cast<Stud>(Ans[i]).grades << static_cast<Stud>(Ans[i]).average << std::endl;
	}
	out << std::endl;
	out.close();
}

// Pagrindin� programa
int main()
{
	thrust::host_vector<Stud> data;
	thrust::device_vector<Stud> dev_data;
	//thrust::device_vector<Stud> rez_temp(arrayCount);
	thrust::device_vector<Stud> dev_rez(inArrayCount);
	//dev_rez[0] = rez_temp[2];
	data = Read("IFF68_LaurinaitisTadas_L4.txt");
	//int i = 0;
	dev_data = data;
	//copy_if(dev_data.begin + i, dev_data.end, rez_temp, is_good());

	int begin = 0;
	for (int i = 0; i < inArrayCount; i++)//11
	{
		thrust::device_vector<Stud> dev_temp;
		for (int j = 0; j < inArrayCount * arrayCount; j++)//55
		{
			if ((j + begin) % inArrayCount == 0) {
				Stud tmp = static_cast<Stud>(dev_data[j]);
				dev_temp.push_back(tmp);
			}
		}
		Stud temp;
		dev_rez[i] = thrust::reduce(dev_temp.begin(), dev_temp.end(), temp, sum_func());

		begin--;
	}
	/*for (int i = 0; i < inArrayCount; i++)
	{
		std::cout << static_cast<Stud>(dev_rez[i]).name << static_cast<Stud>(dev_rez[i]).grades << static_cast<Stud>(dev_rez[i]).average << std::endl;
	}*/
	thrust::host_vector<Stud> rez;
	rez = dev_rez;
	Write("IFF68_LaurinaitisTadas_L4_rez.txt", data, rez);
	return 0;
}